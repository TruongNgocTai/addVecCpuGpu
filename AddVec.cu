#include "hip/hip_runtime.h"
//2nd introduction to programming in cuda 

_global_ void add2Vector(int N, float *d_A, float *d_B, float *d_C)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < N)
		d_C[i] = d_A[i] + d_B[i];
		//cout<<d_C[i];
}

int main(int argc, char **argv)
{
	int N = 5;
	float *A, *B, *C;
	A = (float*)malloc(N*sizeof(float));
	B = (float*)malloc(N*sizeof(float));
	C = (float*)malloc(N*sizeof(float));
	
	for (int i = 0; i < N; i++)
	{
		cout<<"Nhap a"<<i<<": ";
		cin>>A[i];
		cout<<"Nhap b"<<i<<": ";
		cin>>B[i];
	}
	
	for(int j = 0; j < N; j++)
	{
		cout<<a[i]<<"  "<<B[i]<<endl;
	}
	
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, N * sizeof(float));
	hipMalloc(&d_B, N * sizeof(float));
	hipMalloc(&d_C, N * sizeof(float));
	
	hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice);

	add2Vector<<ceil(N/256.0), 256>>(N, d_A, d_B, d_C);
	
	hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost);
	
	cout<<"Ket qua: "<<endl;
	for(int i = 0; i < N; i++)
		cout<<C[i]<< " ";
	hipDeviceReset();
	return 0;
}