#include "hip/hip_runtime.h"
//2nd introduction to programming in cuda 
#include <stdio.h>

// check err
#define CUDA_CHECK(call)\
{\
	hipError_t err = call;\
	if (err != hipSuccess)\
	{\
		printf("%s in %s at line %d!\n", hipGetErrorString(err), __FILE__, __LINE__);\
		exit(EXIT_FAILURE);\
	}\
}

// kernel
__global__ void add2Vector(int N, float *d_A, float *d_B, float *d_C)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < N)
		d_C[i] = d_A[i] + d_B[i];
		//printf("%f  ", d_C[i]);
		//cout<<d_C[i];
}

int main(int argc, char **argv)
{
	int N = 200000000;
	float *A, *B, *C;
	A = (float*)malloc(N*sizeof(float));
	B = (float*)malloc(N*sizeof(float));
	C = (float*)malloc(N*sizeof(float));
	
	for (int i = 0; i < N; i++)
	{
		//printf("Nhap A%d : ", i);
		//scanf("%f", &A[i]);
		//printf("Nhap B%d : ", i);
		//scanf("%f", &B[i]);
		A[i] = i;
		B[i] = i;
	}
	
	
	//for(int j = 0; j < N; j++)
	//{
		//printf("%f   %f\n", A[j], B[j]); 
	//}
	
	
	float *d_A, *d_B, *d_C;
	
	CUDA_CHECK(hipMalloc(&d_A, N * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_B, N * sizeof(float)));
	CUDA_CHECK(hipMalloc(&d_C, N * sizeof(float)));
	
	CUDA_CHECK(hipMemcpy(d_A, A, N*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_B, B, N*sizeof(float), hipMemcpyHostToDevice));
	
	dim3 blockSize(256);
	dim3 gridSize((N-1)/256 + 1);
	
	clock_t begin = clock();
	add2Vector<<<gridSize, blockSize>>>(N, d_A, d_B, d_C);
	
	hipDeviceSynchronize();
	clock_t end = clock();
	CUDA_CHECK(hipGetLastError());
	
	CUDA_CHECK(hipMemcpy(C, d_C, N*sizeof(float), hipMemcpyDeviceToHost));
	
	//printf("Ket qua: \n");
	//for(int i = 0; i < N; i++)
		//printf("%f  ", C[i]);
	float time = (float)(end - begin)/CLOCKS_PER_SEC;
	printf("\nThoi gian tinh toan: %f\n",  time);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipDeviceReset();
	return 0;
}